#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
// Assume the existence of utility functions and CUDA kernel prototypes:
// loadWeights, convLayerKernel, reluKernel, maxPoolingKernel, fullyConnectedKernel
__global__ void conv3DKernelWithBias(float* input, float* output, float* kernel, float* biases,
                                                        int inputHeight, int inputWidth, int inputChannels_l1,
                                                        int outputHeight, int outputWidth, int outputChannels_l1,
                                                        int kernelHeight, int kernelWidth) {
    int outX = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = blockIdx.y * blockDim.y + threadIdx.y;
    int outZ = blockIdx.z * blockDim.z + threadIdx.z;

    if (outX < outputWidth && outY < outputHeight && outZ < outputChannels_l1) {
        float value = 0.0f;

        for (int k = 0; k < inputChannels_l1; ++k) {
            for (int offsetY = 0; offsetY < kernelHeight; ++offsetY) {
                for (int offsetX = 0; offsetX < kernelWidth; ++offsetX) {
                    int inX = outX + offsetX ; // Not Assuming kernel is centered over the pixel
                    int inY = outY + offsetY ;

                    if (inX >= 0 && inX < inputWidth && inY >= 0 && inY < inputHeight) {
                        // Adjust index for channel-consecutive storage
                        int inputIndex = (k * inputHeight * inputWidth) + (inY * inputWidth + inX);
                        
                        int kernelIndex = (outZ * inputChannels_l1 * kernelHeight * kernelWidth) + 
                                          (k * kernelHeight * kernelWidth) + 
                                          (offsetY * kernelWidth) + offsetX;
                        
                        value += input[inputIndex] * kernel[kernelIndex];
                    }
                }
            }
        }

        // Add bias and write to output, adjusting index for channel-consecutive storage
        int outputIndex = (outZ * outputHeight * outputWidth) + (outY * outputWidth + outX);
        output[outputIndex] = value + biases[outZ];
    }
}

__global__ void maxPoolingKernel(float* input, float* output, 
                                                    int inputHeight, int channels,
                                                    int poolHeight, 
                                                    int outputHeight,  int stride) {
    int outX = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = blockIdx.z * blockDim.z + threadIdx.z;

    if (outX < outputHeight && outY < outputHeight && channel < channels) {
        float maxVal = -FLT_MAX;  // Initialize to smallest float value
        for (int poolY = 0; poolY < poolHeight; ++poolY) {
            for (int poolX = 0; poolX < poolHeight; ++poolX) {
                int inX = outX * stride + poolX;
                int inY = outY * stride + poolY;
                if (inX < poolHeight && inY < inputHeight) {
                    // Calculate index for channel-consecutive input
                    int inputIndex = (channel * inputHeight * poolHeight) + (inY * poolHeight + inX);
                    maxVal = fmaxf(maxVal, input[inputIndex]);
                }
            }
        }

        // Write the max value to output, using channel-consecutive indexing
        int outputIndex = (channel * outputHeight * outputHeight) + (outY * outputHeight + outX);
        output[outputIndex] = maxVal;
    }
}

__global__ void reluKernel(float* inputOutput, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        inputOutput[idx] = fmaxf(0.0f, inputOutput[idx]);
    }
}



int main() {
    // Initialize dimensions based on the architecture
    // const int inputSize = 28 * 28;  // Input image size for MNIST
    // const int conv1OutputSize = 24 * 24 * 20; // Output size after first Conv layer
    // Similarly, calculate for other layers...

    // Allocate memory for input and output of each layer, weights, and biases
    // For simplicity, we're using single float pointers; in practice, consider using structures or classes for managing layers
    // float *d_input, *d_conv1Output, *d_conv1Weights, /* other layer outputs, weights, biases */;

    // Allocate GPU memory (hipMalloc) for inputs, outputs, weights, biases

    // Load weights from files and transfer them to GPU memory (hipMemcpy)

    // Setup kernel execution parameters (dim3 blockDim, dim3 gridDim) for each layer

    /// Starting by trying one convolution layer
    // Open the file
    std::ifstream file("./trained_weights/conv1.txt");
    std::ifstream img_file("./output.txt");
    
    if (!file.is_open() || !img_file.is_open()) {
        std::cerr << "Error: Unable to open file" << std::endl;
        return 1;
    }
    
    // Read the values into a vector
    std::vector<float> weights_and_bias;
    std::vector<float> img_dat;
    float value;
    
    while (img_file >> value) {
        img_dat.push_back(value);
    }
    while (file >> value) {
        weights_and_bias.push_back(value);
    }
    
    file.close();
    img_file.close();

     // *** Execute Network Layers ***

    // Convolution Layer 1
    // convLayerKernel<<<gridDimConv1, blockDimConv1>>>(d_input, d_conv1Output, d_conv1Weights, /* other params */);
    // Apply ReLU Activation
    // reluKernel<<<gridDimRelu1, blockDimRelu1>>>(d_conv1Output, d_conv1Output, conv1OutputSize);

    
    // Assuming input image dimensions and Conv_1 output dimensions
    const int inputWidth = 28;
    const int inputHeight = 28;
    const int inputChannels_l1 = 1; // Grayscale image
    const int outputWidth = 24;
    const int outputHeight = 24;
    const int outputChannels_l1 = 20; // Number of filters
    const int kernelHeight = 5;
    const int kernelWidth = 5;

    // Flatten input and output dimensions for easier memory allocation
    const int inputSize = inputWidth * inputHeight * inputChannels_l1;
    const int outputSize = outputWidth * outputHeight * outputChannels_l1;

    float *d_input, *d_output, *d_weights, *d_biases;

    // Allocate device memory
    hipMalloc(&d_input, inputSize * sizeof(float));
    hipMalloc(&d_output, outputSize * sizeof(float));
    // For weights: 20 filters each of size 5x5, and 20 bias values
    hipMalloc(&d_weights, outputChannels_l1 * kernelWidth * kernelHeight * sizeof(float));
    hipMalloc(&d_biases, outputChannels_l1 * sizeof(float));


    hipMemcpy(d_input, img_dat.data() , inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights_and_bias.data(), outputChannels_l1 * kernelWidth * kernelHeight * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, weights_and_bias.data() + outputChannels_l1 * kernelWidth * kernelHeight, outputChannels_l1 * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockDim(16, 16, 1); // Keep z-dimension as 1 for simplicity in 2D convolutions

    dim3 gridDim((outputWidth + blockDim.x - 1) / blockDim.x,
                (outputHeight + blockDim.y - 1) / blockDim.y,
                outputChannels_l1); // Ensure each output channel is handled
    // Launch the convolution kernel
    conv3DKernelWithBias<<<gridDim, blockDim>>>(d_input, d_output, d_weights, d_biases,
                                      inputHeight,  inputWidth,  inputChannels_l1,
                                      outputHeight,  outputWidth,  outputChannels_l1,
                                      kernelHeight,  kernelWidth);
    // Allocate memory for d_output on host
    std::vector<float> h_output(outputWidth * outputHeight * outputChannels_l1);

    // Copy data from device to host
    hipMemcpy(h_output.data(), d_output, outputWidth * outputHeight * outputChannels_l1 * sizeof(float), hipMemcpyDeviceToHost);

    // Output file path
    const std::string output_file_path = "d_output.txt";

    // Open the output file
    std::ofstream output_file(output_file_path);

    if (!output_file.is_open()) {
        std::cerr << "Error: Unable to open output file" << std::endl;
        return 1;
    }

    // Print the d_output to the file
    for (int c = 0; c < outputChannels_l1; ++c) {
        output_file << "Channel " << c << ":" << std::endl;
        for (int i = 0; i < outputHeight; ++i) {
            for (int j = 0; j < outputWidth; ++j) {
                output_file << h_output[(c * outputHeight * outputWidth) + (i * outputWidth) + j] << " ";
            }
            output_file << std::endl;
        }
        output_file << std::endl;
    }

    // Close the output file
    output_file.close();

    std::cout << "d_output written to file: " << output_file_path << std::endl;


    
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_biases);
    // hipFree(d_output); this is input for next layer
   
    // Layer#2 Pooling Layer (Max Pooling)
    // maxPoolingKernel<<<gridDimPool1, blockDimPool1>>>(d_conv1Output, d_pool1Output, /* other params */);
    float *d_output_l2;

    const int poolDimension = 2;
    const int stride = 2;
    const int inputDimension_l2 = 24
    const int outputDimension_l2 = 12;
    const int outputChannels_l2 = 20;
    const int outputSize_l2 = outputDimension_l2*outputDimension_l2*outputChannels_l2;

    hipMalloc(&d_output_l2, outputSize * sizeof(float));

    dim3 blockDim(16, 16, 1); // A common choice, but adjust based on your specific requirements
    dim3 gridDim((outputDimension_l2 + blockDim.x - 1) / blockDim.x, 
                (outputDimension_l2 + blockDim.y - 1) / blockDim.y, 
                outputChannels_l2);

    // Launch the kernel with calculated dimensions
    maxPoolingKernel<<<gridDim, blockDim>>>(d_output, d_output_l2, 
                                                                inputDimension_l2, channels_l2, 
                                                                poolDimension, 
                                                                outputDimension_l2, stride);

    // Layer #3: Convolution Layer
    // Repeat the process for subsequent layers, matching the architecture specifics...
    std::ifstream file2("./trained_weights/conv2.txt");
    
    if (!file2.is_open()) {
        std::cerr << "Error: Unable to open file" << std::endl;
        return 1;
    }
    weights_and_bias.clear();
    
    while (file2 >> value) {
        weights_and_bias.push_back(value);
    }
    // Assuming input image dimensions and Conv_1 output dimensions
    const int inputDimension_l3 = 12;
    const int inputChannels_l3 = 20; // Grayscale image
    const int outputDimension_l3 = 8;
    const int outputChannels_l3 = 50; // Number of filters
    const int kernelDimension_l3 = 5;

    // Flatten input and output dimensions for easier memory allocation
    const int inputSize_l3 = inputDimension_l3 * inputDimension_l3 * inputChannels_l3;
    const int outputSize_l3 = outputDimension_l3 * outputDimension_l3 * outputChannels_l3;

    hipMalloc(&d_output, outputSize * sizeof(float));
    // For weights: 20 filters each of size 5x5, and 20 bias values
    hipMalloc(&d_weights, outputChannels_l3 * kernelDimension_l3 * kernelDimension_l3 * sizeof(float));
    hipMalloc(&d_biases, outputChannels_l3 * sizeof(float));

    hipMemcpy(d_weights, weights_and_bias.data(), outputChannels_l3 * kernelDimension_l3 * kernelDimension_l3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, weights_and_bias.data() + outputChannels_l3 * kernelDimension_l3 * kernelDimension_l3, outputChannels_l3 * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockDim(16, 16, 1); // Keep z-dimension as 1 for simplicity in 2D convolutions

    dim3 gridDim((outputWidth + blockDim.x - 1) / blockDim.x,
                (outputHeight + blockDim.y - 1) / blockDim.y,
                outputChannels_l3); // Ensure each output channel is handled
    // Launch the convolution kernel
    conv3DKernelWithBias<<<gridDim, blockDim>>>(d_output_l2, d_output_l3, d_weights, d_biases,
                                      inputDimension_l3,  inputDimension_l3,  inputChannels_l3,
                                      outputDimension_l3,  outputDimension_l3,  outputChannels_l3,
                                      kernelDimension_l3,  kernelDimension_l3);

    // Layer #4: Maxpooling
    float *d_output_l4;
    const int inputDimension_l4 = 8
    const int outputDimension_l4 = 4;
    const int outputChannels_l4 = 50;
    const int outputSize_l2 = outputDimension_l4*outputDimension_l4*outputChannels_l4;

    hipMalloc(&d_output_l4, outputSize * sizeof(float));

    dim3 blockDim(16, 16, 1); // A common choice, but adjust based on your specific requirements
    dim3 gridDim((outputDimension_l4 + blockDim.x - 1) / blockDim.x, 
                (outputDimension_l4 + blockDim.y - 1) / blockDim.y, 
                outputChannels_l4);

    // Launch the kernel with calculated dimensions
    maxPoolingKernel<<<gridDim, blockDim>>>(d_output_l3, d_output_l4, 
                                                                inputDimension_l4, channels_l4, 
                                                                poolDimension, 
                                                                outputDimension_l4, stride);

    // Fully Connected Layers - Consider handling these differently as they may not be direct convolutions
    // fullyConnectedKernel<<<gridDimFC1, blockDimFC1>>>(d_pool2Output, d_fc1Output, d_fc1Weights, /* other params */);
    // Apply ReLU for FC1

    // FC2 - Output Layer
    // Similar to FC1, adjust for the final output size and activation (if any)

    // Copy final layer output back to host
    // std::vector<float> h_output( /* size of the final output layer */ );
    // hipMemcpy from d_fc2Output to h_output.data()

    // Apply Softmax on CPU for classification
    // std::vector<float> probabilities = softmax(h_output);  // Assuming softmax is defined on the host

    // Output the top probabilities or classification result

    // Cleanup: Free all allocated GPU memory

    return 0;
}
